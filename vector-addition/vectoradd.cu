// Cuda programming practice: Vector addition


#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 8192

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	int i = threadIdx.x;
	if (i<n)
		c[i] = a[i] + b[i];
}

int main()
{
	int *a, *b, *c;
	// Allocate cuda memory
	hipMallocManaged(&a, SIZE * sizeof(int));
	hipMallocManaged(&b, SIZE * sizeof(int));
	hipMallocManaged(&c, SIZE * sizeof(int));
	
	// Initialize the arrays
	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}
	
	// Call addition operation
	VectorAdd <<<1, SIZE>>> (a, b, c, SIZE);

	// Wait for async ops to complete
	hipDeviceSynchronize();

	for (int i = 0; i < 100; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	// Free GPU memory
	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}
